#include "hip/hip_runtime.h"
#if defined(cl_khr_fp64)
#pragma OPENCL EXTENSION cl_khr_fp64: enable
#elif defined(cl_amd_fp64)
#pragma OPENCL EXTENSION cl_amd_fp64: enable
#endif
struct CUDAPrim_Struct
{
	double LocX;
	double LocY;
	double Mass;
	double SpeedX;
	double SpeedY;
	double ForceX;
	double ForceY;
};

struct CUDADebug_Struct
{
	int UB;
	int LB;
	int Other;
};


// Physics_Engine.CUDA
__kernel  void CalcPhysics(global struct CUDAPrim_Struct* Body, int BodyLen0, int nBodies, int RunThreads, global struct CUDAPrim_Struct* OutBody, int OutBodyLen0, global struct CUDADebug_Struct* DebugStuff, int DebugStuffLen0);

// Physics_Engine.CUDA
__kernel  void CalcPhysics(global struct CUDAPrim_Struct* Body, int BodyLen0, int nBodies, int RunThreads, global struct CUDAPrim_Struct* OutBody, int OutBodyLen0, global struct CUDADebug_Struct* DebugStuff, int DebugStuffLen0)
{
	int num = (int)rint((double)nBodies / (double)RunThreads);
	int num2 = nBodies - num * RunThreads;
	int x = get_local_id(0);
	bool flag = x == 0;
	int num3;
	int num4;
	if (flag)
	{
		num3 = 0;
		num4 = num;
	}
	else
	{
		num3 = num * (x - 1) + 1;
		num4 = num * x;
		bool flag2 = x == RunThreads;
		if (flag2)
		{
			num4 += num2;
		}
	}
	double num5 = 0.03;
	double num6 = 1.02;
	int num7 = num3;
	int num8 = num4;
	for (int i = num7; i <= num8; i++)
	{
		Body[(i)].ForceX = 0.0;
		Body[(i)].ForceY = 0.0;
		for (int j = 1; j <= nBodies; j++)
		{
			bool flag3 = i != j;
			if (flag3)
			{
				double num9 = Body[(j)].LocX - OutBody[(i)].LocX;
				double num10 = Body[(j)].LocY - OutBody[(i)].LocY;
				double d = num9 * num9 + num10 * num10;
				double num11 = sqrt(d);
				bool flag4 = num11 > 0.0;
				if (flag4)
				{
					double mass = OutBody[(i)].Mass;
					double mass2 = Body[(j)].Mass;
					double num12 = mass * mass2;
					double num13 = num12 / (num11 * num11 + num6 * num6);
					double num14 = num13 * num9 / num11;
					double num15 = num13 * num10 / num11;
					int expr_17C_cp_0_cp_1 = i;
					OutBody[(expr_17C_cp_0_cp_1)].ForceX = OutBody[(expr_17C_cp_0_cp_1)].ForceX + num14;
					int expr_194_cp_0_cp_1 = i;
					OutBody[(expr_194_cp_0_cp_1)].ForceY = OutBody[(expr_194_cp_0_cp_1)].ForceY + num15;
				}
			}
		}
	}
	int num16 = num3;
	int num17 = num4;
	for (int k = num16; k <= num17; k++)
	{
		int expr_1E0_cp_0_cp_1 = k;
		OutBody[(expr_1E0_cp_0_cp_1)].SpeedX = OutBody[(expr_1E0_cp_0_cp_1)].SpeedX + num5 * OutBody[(k)].ForceX / OutBody[(k)].Mass;
		int expr_216_cp_0_cp_1 = k;
		OutBody[(expr_216_cp_0_cp_1)].SpeedY = OutBody[(expr_216_cp_0_cp_1)].SpeedY + num5 * OutBody[(k)].ForceY / OutBody[(k)].Mass;
		int expr_24C_cp_0_cp_1 = k;
		OutBody[(expr_24C_cp_0_cp_1)].LocX = OutBody[(expr_24C_cp_0_cp_1)].LocX + num5 * OutBody[(k)].SpeedX;
		int expr_273_cp_0_cp_1 = k;
		OutBody[(expr_273_cp_0_cp_1)].LocY = OutBody[(expr_273_cp_0_cp_1)].LocY + num5 * OutBody[(k)].SpeedY;
	}
	DebugStuff[(get_local_id(0))].LB = num3;
        DebugStuff[(get_local_id(0))].UB = num4;
}
