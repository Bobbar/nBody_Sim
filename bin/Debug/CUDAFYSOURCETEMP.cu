
#include <hip/hip_runtime.h>
struct CUDAPrim_Struct
{
	__device__  CUDAPrim_Struct()
	{
	}
	double LocX;
	double LocY;
	float Mass;
	float SpeedX;
	float SpeedY;
	float ForceX;
	float ForceY;
	float ForceTot;
	int Color;
	double Size;
	int Visible;
	int InRoche;
	int BlackHole;
	long long UID;
	int ThreadID;
	int BlockID;
	int BlockDIM;
	int LastColID;
};


// Physics_Engine.CUDA
extern "C" __global__  void CalcPhysics( CUDAPrim_Struct* Body, int BodyLen0, float TimeStep,  CUDAPrim_Struct* OutBody, int OutBodyLen0);
// Physics_Engine.CUDA
extern "C" __global__  void CollideBodies( CUDAPrim_Struct* Body, int BodyLen0,  CUDAPrim_Struct* ColBody, int ColBodyLen0, float TimeStep);

// Physics_Engine.CUDA
extern "C" __global__  void CalcPhysics( CUDAPrim_Struct* Body, int BodyLen0, float TimeStep,  CUDAPrim_Struct* OutBody, int OutBodyLen0)
{
	int num = blockDim.x * blockIdx.x + threadIdx.x;
	float num2 = 2.0f;
	bool flag = num <= BodyLen0 - 1;
	if (flag)
	{
		OutBody[(num)] = Body[(num)];
		bool flag2 = Body[(num)].Visible == 1;
		if (flag2)
		{
			OutBody[(num)].ForceX = 0.0f;
			OutBody[(num)].ForceY = 0.0f;
			OutBody[(num)].ForceTot = 0.0f;
			int num3 = BodyLen0 - 1;
			for (int i = 0; i <= num3; i++)
			{
				bool flag3 = num != i & Body[(i)].Visible == 1;
				if (flag3)
				{
					float num4 = (float)(Body[(i)].LocX - OutBody[(num)].LocX);
					float num5 = (float)(Body[(i)].LocY - OutBody[(num)].LocY);
					float num6 = num4 * num4 + num5 * num5;
					float num7 = (float)sqrt((double)num6);
					bool flag4 = num7 > 0.0f;
					if (flag4)
					{
						float mass = OutBody[(num)].Mass;
						float mass2 = Body[(i)].Mass;
						float num8 = mass * mass2;
						float num9 = num8 / (num7 * num7 + num2 * num2);
						float num10 = num9 * num4 / num7;
						float num11 = num9 * num5 / num7;
						int expr_17C_cp_0_cp_1 = num;
						OutBody[(expr_17C_cp_0_cp_1)].ForceTot = OutBody[(expr_17C_cp_0_cp_1)].ForceTot + num9;
						int expr_191_cp_0_cp_1 = num;
						OutBody[(expr_191_cp_0_cp_1)].ForceX = OutBody[(expr_191_cp_0_cp_1)].ForceX + num10;
						int expr_1A6_cp_0_cp_1 = num;
						OutBody[(expr_1A6_cp_0_cp_1)].ForceY = OutBody[(expr_1A6_cp_0_cp_1)].ForceY + num11;
					}
				}
			}
			bool flag5 = OutBody[(num)].ForceTot > OutBody[(num)].Mass * 4.0f & OutBody[(num)].BlackHole == 0;
			if (flag5)
			{
				OutBody[(num)].InRoche = 1;
			}
			else
			{
				bool flag6 = OutBody[(num)].ForceTot * 2.0f < OutBody[(num)].Mass * 4.0f;
				if (flag6)
				{
					OutBody[(num)].InRoche = 0;
				}
				else
				{
					bool flag7 = OutBody[(num)].BlackHole == 2;
					if (flag7)
					{
						OutBody[(num)].InRoche = 1;
					}
				}
			}
		}
	}
}
// Physics_Engine.CUDA
extern "C" __global__  void CollideBodies( CUDAPrim_Struct* Body, int BodyLen0,  CUDAPrim_Struct* ColBody, int ColBodyLen0, float TimeStep)
{
	int num = blockDim.x * blockIdx.x + threadIdx.x;
	bool flag = num <= BodyLen0 - 1 & Body[(num)].Visible == 1;
	if (flag)
	{
		ColBody[(num)] = Body[(num)];
		int num2 = BodyLen0 - 1;
		for (int i = 0; i <= num2; i++)
		{
			bool flag2 = num != i & Body[(i)].Visible == 1;
			if (flag2)
			{
				float num3 = (float)(Body[(i)].LocX - ColBody[(num)].LocX);
				float num4 = (float)(Body[(i)].LocY - ColBody[(num)].LocY);
				float num5 = num3 * num3 + num4 * num4;
				float num6 = (float)sqrt((double)num5);
				bool flag3 = (double)num6 <= ColBody[(num)].Size / 2.0 + Body[(i)].Size / 2.0;
				if (flag3)
				{
					ColBody[(num)].LastColID = i;
					bool flag4 = num6 > 0.0f;
					if (flag4)
					{
						float speedX = ColBody[(num)].SpeedX;
						float speedY = ColBody[(num)].SpeedY;
						float speedX2 = Body[(i)].SpeedX;
						float speedY2 = Body[(i)].SpeedY;
						float mass = ColBody[(num)].Mass;
						float mass2 = Body[(i)].Mass;
						float num7 = num3 / 2.0f;
						float num8 = num4 / 2.0f;
						num7 /= num6 / 2.0f;
						num8 /= num6 / 2.0f;
						float num9 = num7 * speedX + num8 * speedY;
						float num10 = num7 * speedX2 + num8 * speedY2;
						float num11 = (mass * num9 + mass2 * num10 - mass2 * (num9 - num10)) / (mass + mass2);
						float num12 = (mass * num9 + mass2 * num10 - mass * (num10 - num9)) / (mass + mass2);
						bool flag5 = ColBody[(num)].InRoche == 0 & Body[(i)].InRoche == 1;
						if (flag5)
						{
							bool flag6 = ColBody[(num)].Mass > Body[(i)].Mass;
							if (flag6)
							{
								float speedX3 = ColBody[(num)].SpeedX;
								float speedY3 = ColBody[(num)].SpeedY;
								ColBody[(num)].SpeedX = ColBody[(num)].SpeedX + (num11 - num9) * num7;
								ColBody[(num)].SpeedY = ColBody[(num)].SpeedY + (num11 - num9) * num8;
								float num13 = (float)(3.14159265358979 * pow(ColBody[(num)].Size, 2.0));
								float num14 = (float)(3.14159265358979 * pow(Body[(i)].Size, 2.0));
								num13 += num14;
								ColBody[(num)].Size = sqrt((double)num13 / 3.14159265358979);
								ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
							}
							else
							{
								bool flag7 = ColBody[(num)].Mass == Body[(i)].Mass;
								if (flag7)
								{
									bool flag8 = ColBody[(num)].UID > Body[(i)].UID;
									if (flag8)
									{
										float speedX3 = ColBody[(num)].SpeedX;
										float speedY3 = ColBody[(num)].SpeedY;
										ColBody[(num)].SpeedX = ColBody[(num)].SpeedX + (num11 - num9) * num7;
										ColBody[(num)].SpeedY = ColBody[(num)].SpeedY + (num11 - num9) * num8;
										float num13 = (float)(3.14159265358979 * pow(ColBody[(num)].Size, 2.0));
										float num14 = (float)(3.14159265358979 * pow(Body[(i)].Size, 2.0));
										num13 += num14;
										ColBody[(num)].Size = sqrt((double)num13 / 3.14159265358979);
										ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
									}
									else
									{
										ColBody[(num)].Visible = 0;
									}
								}
								else
								{
									ColBody[(num)].Visible = 0;
								}
							}
						}
						else
						{
							bool flag9 = ColBody[(num)].InRoche == 0 & Body[(i)].InRoche == 0;
							if (flag9)
							{
								bool flag10 = ColBody[(num)].Mass > Body[(i)].Mass;
								if (flag10)
								{
									float speedX3 = ColBody[(num)].SpeedX;
									float speedY3 = ColBody[(num)].SpeedY;
									ColBody[(num)].SpeedX = ColBody[(num)].SpeedX + (num11 - num9) * num7;
									ColBody[(num)].SpeedY = ColBody[(num)].SpeedY + (num11 - num9) * num8;
									float num13 = (float)(3.14159265358979 * pow(ColBody[(num)].Size, 2.0));
									float num14 = (float)(3.14159265358979 * pow(Body[(i)].Size, 2.0));
									num13 += num14;
									ColBody[(num)].Size = sqrt((double)num13 / 3.14159265358979);
									ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
								}
								else
								{
									bool flag11 = ColBody[(num)].Mass == Body[(i)].Mass;
									if (flag11)
									{
										bool flag12 = ColBody[(num)].UID > Body[(i)].UID;
										if (flag12)
										{
											float speedX3 = ColBody[(num)].SpeedX;
											float speedY3 = ColBody[(num)].SpeedY;
											ColBody[(num)].SpeedX = ColBody[(num)].SpeedX + (num11 - num9) * num7;
											ColBody[(num)].SpeedY = ColBody[(num)].SpeedY + (num11 - num9) * num8;
											float num13 = (float)(3.14159265358979 * pow(ColBody[(num)].Size, 2.0));
											float num14 = (float)(3.14159265358979 * pow(Body[(i)].Size, 2.0));
											num13 += num14;
											ColBody[(num)].Size = sqrt((double)num13 / 3.14159265358979);
											ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
										}
										else
										{
											ColBody[(num)].Visible = 0;
										}
									}
									else
									{
										ColBody[(num)].Visible = 0;
									}
								}
							}
							else
							{
								bool flag13 = ColBody[(num)].InRoche == 1 & Body[(i)].InRoche == 1;
								if (flag13)
								{
									mass = ColBody[(num)].Mass;
									mass2 = Body[(i)].Mass;
									float num15 = mass * mass2;
									float num16 = (float)((double)num15 / ((double)(num6 * num6) + (ColBody[(num)].Size / 2.0 + Body[(i)].Size / 2.0)));
									float num17 = num16 * num3 / num6;
									float num18 = num16 * num4 / num6;
									int num19 = (int)rint(40.0 - sqrt((double)ColBody[(num)].Mass) * 2.0 - (double)(TimeStep * 1000.0f));
									int expr_895_cp_0_cp_1 = num;
									ColBody[(expr_895_cp_0_cp_1)].ForceX = ColBody[(expr_895_cp_0_cp_1)].ForceX - num17 * (float)num19;
									int expr_8B0_cp_0_cp_1 = num;
									ColBody[(expr_8B0_cp_0_cp_1)].ForceY = ColBody[(expr_8B0_cp_0_cp_1)].ForceY - num18 * (float)num19;
									double num20 = 0.5;
									int expr_8D6_cp_0_cp_1 = num;
									ColBody[(expr_8D6_cp_0_cp_1)].SpeedX = (float)((double)ColBody[(expr_8D6_cp_0_cp_1)].SpeedX + (double)((num11 - num9) * num7) * num20);
									int expr_8F8_cp_0_cp_1 = num;
									ColBody[(expr_8F8_cp_0_cp_1)].SpeedY = (float)((double)ColBody[(expr_8F8_cp_0_cp_1)].SpeedY + (double)((num11 - num9) * num8) * num20);
								}
								else
								{
									bool flag14 = ColBody[(num)].InRoche == 1 & Body[(i)].InRoche == 0;
									if (flag14)
									{
										ColBody[(num)].Visible = 0;
									}
								}
							}
						}
					}
					else
					{
						bool flag15 = ColBody[(num)].Mass > Body[(i)].Mass;
						if (flag15)
						{
							float num13 = (float)(3.14159265358979 * pow(ColBody[(num)].Size, 2.0));
							float num14 = (float)(3.14159265358979 * pow(Body[(i)].Size, 2.0));
							num13 += num14;
							ColBody[(num)].Size = sqrt((double)num13 / 3.14159265358979);
							ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
						}
						else
						{
							ColBody[(num)].Visible = 0;
						}
					}
				}
			}
		}
		int expr_A44_cp_0_cp_1 = num;
		ColBody[(expr_A44_cp_0_cp_1)].SpeedX = ColBody[(expr_A44_cp_0_cp_1)].SpeedX + TimeStep * ColBody[(num)].ForceX / ColBody[(num)].Mass;
		int expr_A76_cp_0_cp_1 = num;
		ColBody[(expr_A76_cp_0_cp_1)].SpeedY = ColBody[(expr_A76_cp_0_cp_1)].SpeedY + TimeStep * ColBody[(num)].ForceY / ColBody[(num)].Mass;
		int expr_AA8_cp_0_cp_1 = num;
		ColBody[(expr_AA8_cp_0_cp_1)].LocX = ColBody[(expr_AA8_cp_0_cp_1)].LocX + (double)(TimeStep * ColBody[(num)].SpeedX);
		int expr_ACD_cp_0_cp_1 = num;
		ColBody[(expr_ACD_cp_0_cp_1)].LocY = ColBody[(expr_ACD_cp_0_cp_1)].LocY + (double)(TimeStep * ColBody[(num)].SpeedY);
	}
}
