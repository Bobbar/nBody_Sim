
#include <hip/hip_runtime.h>
struct CUDAPrim_Struct
{
	__device__  CUDAPrim_Struct()
	{
	}
	double LocX;
	double LocY;
	double Mass;
	double SpeedX;
	double SpeedY;
	double ForceX;
	double ForceY;
	double ForceTot;
	int Color;
	double Size;
	int Visible;
	int InRoche;
	int BlackHole;
	long long UID;
	int ThreadID;
	int BlockID;
	int BlockDIM;
	int LastColID;
};


// Physics_Engine.CUDA
extern "C" __global__  void CalcPhysics( CUDAPrim_Struct* Body, int BodyLen0, double TimeStep,  CUDAPrim_Struct* OutBody, int OutBodyLen0);
// Physics_Engine.CUDA
extern "C" __global__  void CollideBodies( CUDAPrim_Struct* Body, int BodyLen0,  CUDAPrim_Struct* ColBody, int ColBodyLen0, double TimeStep);

// Physics_Engine.CUDA
extern "C" __global__  void CalcPhysics( CUDAPrim_Struct* Body, int BodyLen0, double TimeStep,  CUDAPrim_Struct* OutBody, int OutBodyLen0)
{
	int num = blockDim.x * blockIdx.x + threadIdx.x;
	double num2 = 2.0;
	bool flag = num <= BodyLen0 - 1;
	if (flag)
	{
		OutBody[(num)] = Body[(num)];
		bool flag2 = Body[(num)].Visible == 1;
		if (flag2)
		{
			OutBody[(num)].ThreadID = threadIdx.x;
			OutBody[(num)].BlockID = blockIdx.x;
			OutBody[(num)].BlockDIM = blockDim.x;
			OutBody[(num)].ForceX = 0.0;
			OutBody[(num)].ForceY = 0.0;
			OutBody[(num)].ForceTot = 0.0;
			int num3 = BodyLen0 - 1;
			for (int i = 0; i <= num3; i++)
			{
				bool flag3 = num != i & Body[(i)].Visible == 1;
				if (flag3)
				{
					double num4 = Body[(i)].LocX - OutBody[(num)].LocX;
					double num5 = Body[(i)].LocY - OutBody[(num)].LocY;
					double d = num4 * num4 + num5 * num5;
					double num6 = sqrt(d);
					bool flag4 = num6 > 0.0;
					if (flag4)
					{
						double mass = OutBody[(num)].Mass;
						double mass2 = Body[(i)].Mass;
						double num7 = mass * mass2;
						double num8 = num7 / (num6 * num6 + num2 * num2);
						double num9 = num8 * num4 / num6;
						double num10 = num8 * num5 / num6;
						int expr_1D1_cp_0_cp_1 = num;
						OutBody[(expr_1D1_cp_0_cp_1)].ForceTot = OutBody[(expr_1D1_cp_0_cp_1)].ForceTot + num8;
						int expr_1E6_cp_0_cp_1 = num;
						OutBody[(expr_1E6_cp_0_cp_1)].ForceX = OutBody[(expr_1E6_cp_0_cp_1)].ForceX + num9;
						int expr_1FB_cp_0_cp_1 = num;
						OutBody[(expr_1FB_cp_0_cp_1)].ForceY = OutBody[(expr_1FB_cp_0_cp_1)].ForceY + num10;
					}
				}
			}
			__syncthreads();
		}
	}
}
// Physics_Engine.CUDA
extern "C" __global__  void CollideBodies( CUDAPrim_Struct* Body, int BodyLen0,  CUDAPrim_Struct* ColBody, int ColBodyLen0, double TimeStep)
{
	int num = blockDim.x * blockIdx.x + threadIdx.x;
	bool flag = num <= BodyLen0 - 1 & Body[(num)].Visible == 1;
	if (flag)
	{
		ColBody[(num)] = Body[(num)];
		int num2 = BodyLen0 - 1;
		for (int i = 0; i <= num2; i++)
		{
			bool flag2 = num != i & Body[(i)].Visible == 1;
			if (flag2)
			{
				double num3 = Body[(i)].LocX - ColBody[(num)].LocX;
				double num4 = Body[(i)].LocY - ColBody[(num)].LocY;
				double d = num3 * num3 + num4 * num4;
				double num5 = sqrt(d);
				bool flag3 = num5 <= ColBody[(num)].Size / 2.0 + Body[(i)].Size / 2.0;
				if (flag3)
				{
					ColBody[(num)].LastColID = i;
					bool flag4 = num5 > 0.0;
					if (flag4)
					{
						double speedX = ColBody[(num)].SpeedX;
						double speedY = ColBody[(num)].SpeedY;
						double speedX2 = Body[(i)].SpeedX;
						double speedY2 = Body[(i)].SpeedY;
						double mass = ColBody[(num)].Mass;
						double mass2 = Body[(i)].Mass;
						double num6 = num3 / 2.0;
						double num7 = num4 / 2.0;
						num6 /= num5 / 2.0;
						num7 /= num5 / 2.0;
						double num8 = num6 * speedX + num7 * speedY;
						double num9 = num6 * speedX2 + num7 * speedY2;
						double num10 = (mass * num8 + mass2 * num9 - mass2 * (num8 - num9)) / (mass + mass2);
						double num11 = (mass * num8 + mass2 * num9 - mass * (num9 - num8)) / (mass + mass2);
						bool flag5 = ColBody[(num)].InRoche == 0 & Body[(i)].InRoche == 1;
						if (flag5)
						{
							bool flag6 = ColBody[(num)].Mass > Body[(i)].Mass;
							if (flag6)
							{
								double speedX3 = ColBody[(num)].SpeedX;
								double speedY3 = ColBody[(num)].SpeedY;
								ColBody[(num)].SpeedX = ColBody[(num)].SpeedX + (num10 - num8) * num6;
								ColBody[(num)].SpeedY = ColBody[(num)].SpeedY + (num10 - num8) * num7;
								Body[(i)].Visible = 0;
								double num12 = 3.14159265358979 * pow(ColBody[(num)].Size, 2.0);
								double num13 = 3.14159265358979 * pow(Body[(i)].Size, 2.0);
								num12 += num13;
								ColBody[(num)].Size = sqrt(num12 / 3.14159265358979);
								ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
							}
							else
							{
								bool flag7 = ColBody[(num)].Mass == Body[(i)].Mass;
								if (flag7)
								{
									bool flag8 = ColBody[(num)].UID > Body[(i)].UID;
									if (flag8)
									{
										double speedX3 = ColBody[(num)].SpeedX;
										double speedY3 = ColBody[(num)].SpeedY;
										ColBody[(num)].SpeedX = ColBody[(num)].SpeedX + (num10 - num8) * num6;
										ColBody[(num)].SpeedY = ColBody[(num)].SpeedY + (num10 - num8) * num7;
										Body[(i)].Visible = 0;
										double num12 = 3.14159265358979 * pow(ColBody[(num)].Size, 2.0);
										double num13 = 3.14159265358979 * pow(Body[(i)].Size, 2.0);
										num12 += num13;
										ColBody[(num)].Size = sqrt(num12 / 3.14159265358979);
										ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
									}
									else
									{
										ColBody[(num)].Visible = 0;
									}
								}
							}
						}
						else
						{
							bool flag9 = ColBody[(num)].InRoche == 0 & Body[(i)].InRoche == 0;
							if (flag9)
							{
								bool flag10 = ColBody[(num)].Mass > Body[(i)].Mass;
								if (flag10)
								{
									double speedX3 = ColBody[(num)].SpeedX;
									double speedY3 = ColBody[(num)].SpeedY;
									ColBody[(num)].SpeedX = ColBody[(num)].SpeedX + (num10 - num8) * num6;
									ColBody[(num)].SpeedY = ColBody[(num)].SpeedY + (num10 - num8) * num7;
									Body[(i)].Visible = 0;
									double num12 = 3.14159265358979 * pow(ColBody[(num)].Size, 2.0);
									double num13 = 3.14159265358979 * pow(Body[(i)].Size, 2.0);
									num12 += num13;
									ColBody[(num)].Size = sqrt(num12 / 3.14159265358979);
									ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
								}
								else
								{
									bool flag11 = ColBody[(num)].Mass == Body[(i)].Mass;
									if (flag11)
									{
										bool flag12 = ColBody[(num)].UID > Body[(i)].UID;
										if (flag12)
										{
											double speedX3 = ColBody[(num)].SpeedX;
											double speedY3 = ColBody[(num)].SpeedY;
											ColBody[(num)].SpeedX = ColBody[(num)].SpeedX + (num10 - num8) * num6;
											ColBody[(num)].SpeedY = ColBody[(num)].SpeedY + (num10 - num8) * num7;
											Body[(i)].Visible = 0;
											double num12 = 3.14159265358979 * pow(ColBody[(num)].Size, 2.0);
											double num13 = 3.14159265358979 * pow(Body[(i)].Size, 2.0);
											num12 += num13;
											ColBody[(num)].Size = sqrt(num12 / 3.14159265358979);
											ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
										}
										else
										{
											ColBody[(num)].Visible = 0;
										}
									}
									else
									{
										ColBody[(num)].Visible = 0;
									}
								}
							}
							else
							{
								bool flag13 = ColBody[(num)].InRoche == 1 & Body[(i)].InRoche == 1;
								if (flag13)
								{
									mass = ColBody[(num)].Mass;
									mass2 = Body[(i)].Mass;
									double num14 = mass * mass2;
									double num15 = 1.02;
									double num16 = num14 / (num5 * num5 + num15 * num15);
									double num17 = num16 * num3 / num5;
									double num18 = num16 * num4 / num5;
									int num19 = 40;
									int expr_85D_cp_0_cp_1 = num;
									ColBody[(expr_85D_cp_0_cp_1)].ForceX = ColBody[(expr_85D_cp_0_cp_1)].ForceX - num17 * (double)num19;
									int expr_878_cp_0_cp_1 = num;
									ColBody[(expr_878_cp_0_cp_1)].ForceY = ColBody[(expr_878_cp_0_cp_1)].ForceY - num18 * (double)num19;
									double num20 = 0.1;
									int expr_89E_cp_0_cp_1 = num;
									ColBody[(expr_89E_cp_0_cp_1)].SpeedX = ColBody[(expr_89E_cp_0_cp_1)].SpeedX + (num10 - num8) * num6 * num20;
									int expr_8BD_cp_0_cp_1 = num;
									ColBody[(expr_8BD_cp_0_cp_1)].SpeedY = ColBody[(expr_8BD_cp_0_cp_1)].SpeedY + (num10 - num8) * num7 * num20;
								}
								else
								{
									bool flag14 = ColBody[(num)].InRoche == 1 & Body[(i)].InRoche == 0;
									if (flag14)
									{
										ColBody[(num)].Visible = 0;
									}
								}
							}
						}
					}
					else
					{
						bool flag15 = ColBody[(num)].Mass > Body[(i)].Mass;
						if (flag15)
						{
							double num12 = 3.14159265358979 * pow(ColBody[(num)].Size, 2.0);
							double num13 = 3.14159265358979 * pow(Body[(i)].Size, 2.0);
							num12 += num13;
							ColBody[(num)].Size = sqrt(num12 / 3.14159265358979);
							ColBody[(num)].Mass = ColBody[(num)].Mass + Body[(i)].Mass;
						}
						else
						{
							ColBody[(num)].Visible = 0;
						}
					}
				}
			}
		}
		int expr_A03_cp_0_cp_1 = num;
		ColBody[(expr_A03_cp_0_cp_1)].SpeedX = ColBody[(expr_A03_cp_0_cp_1)].SpeedX + TimeStep * ColBody[(num)].ForceX / ColBody[(num)].Mass;
		int expr_A35_cp_0_cp_1 = num;
		ColBody[(expr_A35_cp_0_cp_1)].SpeedY = ColBody[(expr_A35_cp_0_cp_1)].SpeedY + TimeStep * ColBody[(num)].ForceY / ColBody[(num)].Mass;
		int expr_A67_cp_0_cp_1 = num;
		ColBody[(expr_A67_cp_0_cp_1)].LocX = ColBody[(expr_A67_cp_0_cp_1)].LocX + TimeStep * ColBody[(num)].SpeedX;
		int expr_A8B_cp_0_cp_1 = num;
		ColBody[(expr_A8B_cp_0_cp_1)].LocY = ColBody[(expr_A8B_cp_0_cp_1)].LocY + TimeStep * ColBody[(num)].SpeedY;
	}
	__syncthreads();
}
